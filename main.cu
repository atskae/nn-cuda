
#include <hip/hip_runtime.h>
#include <stdio.h>

#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

__global__ void test() {
    printf("Hello from GPU!\n");
}

int main(int argc, char* argv[]) {

    int bx = 2;
    int by = 2;
    int numBlocks = 1;
    dim3 block(bx, by);
    dim3 grid(numBlocks);

    // launch CUDA kernel 
    test<<<grid, block>>>();

    printf("Hello from CPU!\n");

    return 0;
}
