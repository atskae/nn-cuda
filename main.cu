
#include <hip/hip_runtime.h>
#include <stdio.h>

#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

__global__ void test() {
    printf("Hello from GPU!\n");
}

int main(int argc, char* argv[]) {

    if(argc < 2) {
        printf("Usage: ./nn-cuda <mnist path>");
        return 1;
    }    

    int bx = 2;
    int by = 2;
    int numBlocks = 1;
    dim3 block(bx, by);
    dim3 grid(numBlocks);

    // launch CUDA kernel 
    test<<<grid, block>>>();

    printf("Hello from CPU!\n");

    return 0;
}
